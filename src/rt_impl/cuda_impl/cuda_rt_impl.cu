#include "hip/hip_runtime.h"
//
// Created by 榨汁机 on 4/22/2021.
//

#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "include/utils.h"
#include "include/camera.h"
#include "include/scene.h"
#include "include/pixel.h"
#include "include/ray.h"


#define malloc_or_exit(err, var, size) do {                             \
    err = hipMalloc(&var, size);                                       \
    if (err != hipSuccess) {                                           \
        fprintf(stderr, "Failed to allocate var on device.\n");         \
        fprintf(stderr, "Error Code: %s\n", hipGetErrorString(err));   \
        successfully_exit = FALSE;                                      \
        goto cleanup_and_exit;                                          \
    }                                                                   \
} while(0)

#define memcpy_or_exit(err, dst, src, size, kind) do {                  \
    err = hipMemcpy(dst, src, size, kind);                             \
    if (err != hipSuccess) {                                           \
        fprintf(stderr, "Failed to copy memory.\n");                    \
        fprintf(stderr, "Error Code: %s\n", hipGetErrorString(err));   \
        successfully_exit = FALSE;                                      \
        goto cleanup_and_exit;                                          \
    }                                                                   \
} while (0)

#define free_if_is_not_null(ptr) do {   \
    if (ptr != NULL) {                  \
        hipFree(ptr);                  \
    }                                   \
} while(0)

__global__ void calculate_rays(ray_t *rays, camera_t camera) {
    size_t const idx = blockDim.x * blockIdx.x + threadIdx.y;
    ray_t CPTR ray = rays + idx;

    canvas_t CPTRC canvas = &camera.canvas;
    real_t const pix_width = (real_t) canvas->width / canvas->res_x;
    real_t const pix_height = (real_t) canvas->height / canvas->res_y;
    real_t const half_width = canvas->width / 2;
    real_t const half_height = canvas->height / 2;
    size_t const u = idx % canvas->res_x;
    size_t const v = idx / canvas->res_x;

    ray->origin = camera.observer.origin;
    ray->direction.d[0] = 0;
    ray->direction.d[1] = half_width - u * pix_width;
    ray->direction.d[2] = half_height - v * pix_height;
}


int cuda_rt_impl(camera_t CPTRC _camera, scene_t CPTR _scene) {
    hipError_t err;
    bool_t successfully_exit = TRUE;

    size_t s;
    size_t num_blocks_per_grid;
    size_t num_threads_per_block;

    pix_t *d_canvas = NULL;
    patch_t *d_patches = NULL;
    light_t *d_lights = NULL;

    real_t roll, pitch, yaw;
    mat3_t rotation;
    mat3_t *d_rot = NULL;

    size_t num_rays;
    ray_t *d_rays = NULL;

    // allocate canvas on device
    s = _camera->canvas.res_x * _camera->canvas.res_y;  // size of the canvas
    malloc_or_exit(err, d_canvas, s);

    // copy scene to device
    s = _scene->num_patches * sizeof(patch_t);
    malloc_or_exit(err, d_patches, s);
    memcpy_or_exit(err, d_patches, _scene->patches, s, hipMemcpyHostToDevice);
    s = _scene->num_lights * sizeof(light_t);
    malloc_or_exit(err, d_lights, s);
    memcpy_or_exit(err, d_lights, _scene->lights, s, hipMemcpyHostToDevice);

    // calculate rotation matrix, copy it to device and construct cuBLAS matrix
    roll = _camera->observer.orientation.d[0];
    pitch = _camera->observer.orientation.d[1];
    yaw = _camera->observer.orientation.d[2];
    get_rotation_matrix(&rotation, roll, pitch, yaw);
    s = sizeof(mat3_t);
    malloc_or_exit(err, d_rot, s);
    memcpy_or_exit(err, d_rot, &rotation, s, hipMemcpyHostToDevice);

    // allocate rays on device
    num_rays = _camera->canvas.res_x * _camera->canvas.res_y;
    s = num_rays * sizeof(ray_t);
    malloc_or_exit(err, d_rays, s);

    num_threads_per_block = 1024;
    num_blocks_per_grid =
            (num_rays + num_threads_per_block - 1) / num_threads_per_block;
    calculate_rays<<<num_blocks_per_grid, num_threads_per_block>>>(
            d_rays, *_camera
    );

    cleanup_and_exit:
    free_if_is_not_null(d_canvas);
    free_if_is_not_null(d_patches);
    free_if_is_not_null(d_lights);
    free_if_is_not_null(d_rot);
    free_if_is_not_null(d_rays);
    return successfully_exit ? EXIT_SUCCESS : EXIT_FAILURE;
}
